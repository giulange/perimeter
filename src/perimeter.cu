#include "hip/hip_runtime.h"
/**
 * Copyright 2015-2020 GeoProc Service.  All rights reserved.
 *
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation outside the terms of the EULA is strictly prohibited.
 *
 */

//	INCLUDES
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <errno.h>        	/* errno */
#include <string.h>       	/* strerror */
#include <math.h>			// ceil
#include <time.h>			// CLOCKS_PER_SEC

// CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// GIS
#include "/home/giuliano/git/cuda/weatherprog-cudac/includes/gis.h"

/**
 * 	PARS
 */
//static const int WORK_SIZE = 256;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }


/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(void) {
	void *d = NULL;
	int i;
	unsigned int idata[WORK_SIZE], odata[WORK_SIZE];

	for (i = 0; i < WORK_SIZE; i++)
		idata[i] = (unsigned int) i;

	CUDA_CHECK_RETURN(hipMalloc((void**) &d, sizeof(int) * WORK_SIZE));
	CUDA_CHECK_RETURN(
			hipMemcpy(d, idata, sizeof(int) * WORK_SIZE, hipMemcpyHostToDevice));

	bitreverse<<<1, WORK_SIZE, WORK_SIZE * sizeof(int)>>>(d);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError());
	CUDA_CHECK_RETURN(hipMemcpy(odata, d, sizeof(int) * WORK_SIZE, hipMemcpyDeviceToHost));

	for (i = 0; i < WORK_SIZE; i++)
		printf("Input value: %u, device output: %u\n", idata[i], odata[i]);

	CUDA_CHECK_RETURN(hipFree((void*) d));
	CUDA_CHECK_RETURN(hipDeviceReset());

	return 0;
}
